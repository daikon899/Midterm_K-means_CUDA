#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <numeric>
#include <hip/hip_runtime.h>

#include "Point.h"
#include "csvHandler.h"

using namespace std;

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

//--------------------------------------------------------------------------------------------------

int main(int argc, char **argv)
{
	initialize();
	vector<Point> data = readCsv();
	//kMeans(&data, 500, k);

}
