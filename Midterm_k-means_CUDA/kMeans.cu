#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <numeric>
#include <hip/hip_runtime.h>
#include <random>

#include "csvHandler.h"
#include "definitions.h"

#include <unistd.h>
#include <chrono>
using namespace std::chrono;

using namespace std;

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

//---------------------------------------------------------------------------------------------------------------------------------------

__global__ void warm_up_gpu(){  // this kernel avoids cold start when evaluating duration of kmeans exec.
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid;
}

//----------------------------------------------------------------------------------------------------------------------------------------

__global__ void updateCentroids(float *points_d, float *centroids_d, int *assignedCentroids_d, int *numPoints_d, int numDataset){
 // more parallelizable
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float sums_s[CLUSTER_NUM * 3];
	__shared__ int numPoints_s[CLUSTER_NUM];
	if(threadIdx.x < CLUSTER_NUM * 3) {
		if(threadIdx.x < CLUSTER_NUM) {
			numPoints_s[threadIdx.x] = 0;
		}
		sums_s[threadIdx.x] = 0.0f;
	}
	__syncthreads();

	if (tid < DATA_SIZE * numDataset){
		int cluster = assignedCentroids_d[tid];
		atomicAdd(&sums_s[cluster * 3], points_d[tid * 3]);
		atomicAdd(&sums_s[cluster * 3 + 1], points_d[tid * 3 + 1]);
		atomicAdd(&sums_s[cluster * 3 + 2], points_d[tid * 3 + 2]);
		atomicAdd(&numPoints_s[cluster], 1);
	}

	__syncthreads();

	//commit to global memory
	if(threadIdx.x < CLUSTER_NUM * 3) {
		atomicAdd(&centroids_d[threadIdx.x], sums_s[threadIdx.x]);
		if(threadIdx.x < CLUSTER_NUM) {
			atomicAdd(&numPoints_d[threadIdx.x], numPoints_s[threadIdx.x]);
		}
	}

}

__global__ void calculateMeans(float *centroids_d, int *numPoints_d){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid < CLUSTER_NUM * 3){
			centroids_d[tid] = centroids_d[tid] / numPoints_d[tid / 3];
	}
}


__global__ void assignClusters(float *points_d, float *centroids_d, int *assignedCentroids_d, bool *clusterChanged, int numDataset){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float centroids_s[CLUSTER_NUM * 3];
	if (threadIdx.x < CLUSTER_NUM * 3) {
		centroids_s[threadIdx.x] = centroids_d[threadIdx.x];
	}

	if(tid < DATA_SIZE * numDataset) {
		float clusterDistance = __FLT_MAX__;
		int oldCluster = assignedCentroids_d[tid];
		int currentCluster = oldCluster;
		float pX = points_d[tid * 3];
		float pY = points_d[tid * 3 + 1];
		float pZ = points_d[tid * 3 + 2];


		for (int j = 0; j < CLUSTER_NUM; j++) {
			float distanceX = centroids_s[j * 3] - pX;
			float distanceY = centroids_s[j * 3 + 1] - pY;
			float distanceZ = centroids_s[j * 3 + 2] - pZ;
			float distance = sqrt(pow(distanceX, 2) + pow(distanceY, 2) + pow(distanceZ, 2));
			if (distance < clusterDistance) {
				clusterDistance = distance;
				currentCluster = j;
			}
		}

		if (currentCluster != oldCluster) {
		   *clusterChanged = true;
		   assignedCentroids_d[tid] = currentCluster;
		}
	}

}

__host__ void kMeansCuda(float *points_h, int epochsLimit, int numDataset){
	//device memory managing
	float *points_d, *centroids_d;
	int *assignedCentroids_d, *numPoints_d;
	int *assignedCentroids_h = (int*) malloc(sizeof(int) * DATA_SIZE * numDataset);
	CUDA_CHECK_RETURN(hipMalloc((void ** )&points_d, sizeof(float) * DATA_SIZE * numDataset * 3));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&centroids_d, sizeof(float) * CLUSTER_NUM * 3));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&assignedCentroids_d, sizeof(int) * DATA_SIZE * numDataset));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&numPoints_d, sizeof(int) * CLUSTER_NUM ));

	CUDA_CHECK_RETURN(hipMemcpy(points_d, points_h, sizeof(float) * DATA_SIZE * numDataset * 3, hipMemcpyHostToDevice)); 

	// Step 1: Create k random centroids
	float *centroids_h = (float*) malloc(sizeof(float) * CLUSTER_NUM * 3); 
	//srand(time(NULL));
	//int randNum = 5;
	//int randNum = rand() % ((DATA_SIZE * numDataset) / CLUSTER_NUM);
	random_device rd;
	default_random_engine engine(rd());
	uniform_int_distribution<int> distribution(0, DATA_SIZE * numDataset - 1);
	for (int i = 0; i < CLUSTER_NUM; i++){
		int randomLocation = distribution(engine);
		//int randomLocation = randNum + (DATA_SIZE * numDataset) * i / CLUSTER_NUM;
		centroids_h[i * 3] = points_h[randomLocation  * 3];
		centroids_h[i * 3 + 1] = points_h[randomLocation * 3 + 1];
		centroids_h[i * 3 + 2] = points_h[randomLocation * 3 + 2];
	}

	CUDA_CHECK_RETURN(hipMemcpy(centroids_d, centroids_h, sizeof(float) * CLUSTER_NUM * 3, hipMemcpyHostToDevice));

	bool clusterChanged_h = false;
	bool *ptrCgd_h = &clusterChanged_h;
	bool *clusterChanged_d;
	CUDA_CHECK_RETURN(hipMalloc(&clusterChanged_d, sizeof(bool)));



	int epoch = 0;
	while(epoch < epochsLimit) {
		//Step 2: assign dataPoints to the clusters, based on the distance from its centroid

		CUDA_CHECK_RETURN(hipMemcpy(clusterChanged_d, ptrCgd_h, sizeof(bool), hipMemcpyHostToDevice));
		assignClusters<<<(DATA_SIZE * numDataset + 127)/ 128 , 128>>>(points_d, centroids_d, assignedCentroids_d, clusterChanged_d, numDataset);
		hipDeviceSynchronize();

		//write a csv file at each iteration to check how k-means is assigning clusters
		//CUDA_CHECK_RETURN(hipMemcpy(assignedCentroids_h, assignedCentroids_d, sizeof(int) * DATA_SIZE, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(ptrCgd_h, clusterChanged_d, sizeof(bool), hipMemcpyDeviceToHost));
		//writeCsv(points_h, centroids_h, assignedCentroids_h, epoch);


		if (!clusterChanged_h) {
		    //printf("Nothing changed...exiting \n");
		    break;          // exit if clusters has not been changed
		}
		else { clusterChanged_h = false; }

		//Step 3: update centroids

		// set numPoints_d and centroids_d to 0 so updateCentroids can do is stuff to evaluate the new position of the centroids
		CUDA_CHECK_RETURN(hipMemset(numPoints_d, 0 , sizeof(int) * CLUSTER_NUM));
		CUDA_CHECK_RETURN(hipMemset(centroids_d, 0 , sizeof(float) * CLUSTER_NUM * 3));
		updateCentroids<<<(DATA_SIZE * numDataset + 127) / 128 , 128>>>(points_d, centroids_d, assignedCentroids_d, numPoints_d, numDataset);
		hipDeviceSynchronize();
		calculateMeans<<<(CLUSTER_NUM * 3 + 31) / 32, 32 >>>(centroids_d, numPoints_d);
		hipDeviceSynchronize();
		//CUDA_CHECK_RETURN(hipMemcpy(centroids_h, centroids_d, sizeof(float) * CLUSTER_NUM * 3, hipMemcpyDeviceToHost));  // use it in case you want the code to write the csv's at each iteration


		//printf("iteration %d complete\n", epoch + 1);
		epoch++;
	}


	if (epoch == epochsLimit){
		printf("Maximum number of iterations reached! \n");
	}

	printf("iterations = %d \n", epoch);

	 // Free host memory
	//free(points_h);
	free(centroids_h);
	free(assignedCentroids_h);

	 //free device memory
	hipFree(points_d);
	hipFree(centroids_d);
	hipFree(assignedCentroids_d);
	hipFree(clusterChanged_d);
	hipFree(numPoints_d);
}

int main(int argc, char **argv){

	initialize();
	float *data_h = readCsv(1);
	warm_up_gpu<<<128, 128>>>();  // avoiding cold start...
	auto start = high_resolution_clock::now();
	kMeansCuda(data_h, MAX_ITERATIONS, 1);
	auto end = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(end - start);
	cout<< "duration = " << duration.count() << " microseconds" << endl;

	free(data_h);

}

